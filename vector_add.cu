#include <cstdio>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

// error checking
#define CUDA_OK(expr)                                                         \
  do {                                                                        \
    hipError_t _e = (expr);                                                  \
    if (_e != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error %s at %s:%d\n",                             \
              hipGetErrorString(_e), __FILE__, __LINE__);                    \
      exit(1);                                                                \
    }                                                                         \
  } while (0)


__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    const int n = 1 << 20;              // 1,048,576
    const size_t bytes = n * sizeof(float);

    std::vector<float> ha(n, 1.0f), hb(n, 2.0f), hc(n);
    float *da = nullptr, *db = nullptr, *dc = nullptr;

    // cudaMalloc(&da, bytes);
    // cudaMalloc(&db, bytes);
    // cudaMalloc(&dc, bytes);

    // cudaMemcpy(da, ha.data(), bytes, cudaMemcpyHostToDevice);
    // cudaMemcpy(db, hb.data(), bytes, cudaMemcpyHostToDevice);

    CUDA_OK(hipMalloc(&da, bytes));
    CUDA_OK(hipMalloc(&db, bytes));
    CUDA_OK(hipMalloc(&dc, bytes));

    CUDA_OK(hipMemcpy(da, ha.data(), bytes, hipMemcpyHostToDevice));
    CUDA_OK(hipMemcpy(db, hb.data(), bytes, hipMemcpyHostToDevice));

    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);

    hipEvent_t t0, t1;
    CUDA_OK(hipEventCreate(&t0));
    CUDA_OK(hipEventCreate(&t1));
    CUDA_OK(hipEventRecord(t0));
    vectorAdd<<<grid, block>>>(da, db, dc, n);
    CUDA_OK(hipGetLastError());            // kernel launch status
    CUDA_OK(hipEventRecord(t1));
    CUDA_OK(hipEventSynchronize(t1));      // waits for kernel to finish

    float ms=0.f;
    CUDA_OK(hipEventElapsedTime(&ms, t0, t1));
    printf("vectorAdd: %.3f ms\n", ms);

    CUDA_OK(hipEventDestroy(t0));
    CUDA_OK(hipEventDestroy(t1));

    CUDA_OK(hipMemcpy(hc.data(), dc, bytes, hipMemcpyDeviceToHost));

    // vectorAdd<<<grid, block>>>(da, db, dc, n);
    // cudaDeviceSynchronize();

    // cudaMemcpy(hc.data(), dc, bytes, cudaMemcpyDeviceToHost);

    // quick correctness check
    for (int i = 0; i < 5; ++i) std::printf("c[%d]=%.1f\n", i, hc[i]); // expect 3.0
    float max_err = 0.f;
    for (int i = 0; i < n; ++i) max_err = fmaxf(max_err, fabsf(hc[i] - 3.0f));
    std::printf("max error = %g\n", max_err);

    CUDA_OK(hipFree(da));
    CUDA_OK(hipFree(db));
    CUDA_OK(hipFree(dc));

    // cudaFree(da); cudaFree(db); cudaFree(dc);
    return 0;
}