#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    const int n = 1 << 20;              // 1,048,576
    const size_t bytes = n * sizeof(float);

    std::vector<float> ha(n, 1.0f), hb(n, 2.0f), hc(n);
    float *da = nullptr, *db = nullptr, *dc = nullptr;

    hipMalloc(&da, bytes);
    hipMalloc(&db, bytes);
    hipMalloc(&dc, bytes);

    hipMemcpy(da, ha.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(db, hb.data(), bytes, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);

    vectorAdd<<<grid, block>>>(da, db, dc, n);
    hipDeviceSynchronize();

    hipMemcpy(hc.data(), dc, bytes, hipMemcpyDeviceToHost);

    // quick correctness check
    for (int i = 0; i < 5; ++i) std::printf("c[%d]=%.1f\n", i, hc[i]); // expect 3.0
    float max_err = 0.f;
    for (int i = 0; i < n; ++i) max_err = fmaxf(max_err, fabsf(hc[i] - 3.0f));
    std::printf("max error = %g\n", max_err);

    hipFree(da); hipFree(db); hipFree(dc);
    return 0;
}