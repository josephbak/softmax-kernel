
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void hello() {
    printf("Hello from GPU thread %d, block %d!\n", threadIdx.x, blockIdx.x);
}
int main() {
    hello<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}
